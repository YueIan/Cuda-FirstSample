#include "hip/hip_runtime.h"
#include<math.h>
#include "sobel.cuh"
#include "utils.cuh"

extern bool InitCUDA();

#define MONITOR_TIME
#define NUM_THREADS 256

__global__ void doConvolutionInLineCUDA(
    const short* src,
    short* des,
    const int x,
    const int y,
    const int z
) {
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;
    int i = 0;
    int j = (block_id*NUM_THREADS + thread_id) % y;
    int k = (block_id*NUM_THREADS + thread_id) / y;

    if(j<=0 || j>=y-1 || k <=0 || k>= z-1){
        return;
    }

    short s_x, s_y, s_z;
    short _s_x, _s_y, _s_z;
    float temp;
    short* temp_m = new short[9];
    for(i;i<x;i++){
        if(i<=0||i>=x-1){
            continue;
        }
        
        _s_x = src[(k-1)*y*x+ (j-1)*x + (i-1)] + src[(k-1)*y*x + (j-1)*x + (i)] * 3 + src[(k-1)*y*x + (j-1)*x + (i+1)] +
            src[(k-1)*y*x + (j)*x + (i-1)] *3 + src[(k-1)*y*x + (j)*x + (i)] * 6 + src[(k-1)*y*x + (j)*x + (i+1)] * 3 +
            src[(k-1)*y*x + (j+1)*x + (i-1)] + src[(k-1)*y*x + (j+1)*x + (i)] * 3 + src[(k-1)*y*x + (j+1)*x + (i+1)];

        _s_z = src[(k+1)*y*x + (j-1)*x + (i-1)] + src[(k+1)*y*x + (j-1)*x + (i)] * 3 + src[(k+1)*y*x + (j-1)*x + (i+1)] +
            src[(k+1)*y*x + (j)*x + (i-1)] *3 + src[(k+1)*y*x + (j)*x + (i)] * 6 + src[(k+1)*y*x + (j)*x + (i+1)] * 3 +
            src[(k+1)*y*x + (j+1)*x + (i-1)] + src[(k+1)*y*x + (j+1)*x + (i)] * 3 + src[(k+1)*y*x + (j+1)*x + (i+1)];

        s_x = _s_z - _s_x;

        _s_x = src[(k-1)*y*x + (j-1)*x + (i-1)] + src[(k-1)*y*x + (j-1)*x + (i)] * 3 + src[(k-1)*y*x + (j-1)*x + (i+1)] +
            src[(k-1)*y*x + (j+1)*x + (i-1)] * -1 + src[(k-1)*y*x + (j+1)*x + (i)] * -3 + src[(k-1)*y*x + (j+1)*x + (i+1)] * -1;

        _s_y = src[(k)*y*x + (j-1)*x + (i-1)] * 3 + src[(k)*y*x + (j-1)*x + (i)] * 6 + src[(k)*y*x + (j-1)*x + (i+1)] * 3+
            src[(k)*y*x + (j+1)*x + (i-1)] * -3 + src[(k)*y*x + (j+1)*x + (i)] * -6 + src[(k)*y*x + (j+1)*x + (i+1)] * -3;

        _s_z = src[(k+1)*y*x + (j-1)*x + (i-1)] + src[(k+1)*y*x + (j-1)*x + (i)] * 3 + src[(k+1)*y*x + (j-1)*x + (i+1)] +
            src[(k+1)*y*x + (j+1)*x + (i-1)] * -1 + src[(k+1)*y*x + (j+1)*x + (i)] * -3 + src[(k+1)*y*x + (j+1)*x + (i+1)] * -1;

        s_y = _s_x + _s_y + _s_z;

        _s_x = src[(k-1)*y*x + (j-1)*x + (i-1)] * -1 + src[(k-1)*y*x + (j-1)*x + (i+1)] +
            src[(k-1)*y*x + (j)*x + (i-1)] * -3 + src[(k-1)*y*x + (j)*x + (i+1)] * 3 +
            src[(k-1)*y*x + (j+1)*x + (i-1)] * -1 + src[(k-1)*y*x + (j+1)*x + (i+1)];

        _s_y = src[(k)*y*x + (j-1)*x + (i-1)] * -3 + src[(k)*y*x + (j-1)*x + (i+1)] * 3+
            src[(k)*y*x + (j)*x + (i-1)] * -6 + src[(k)*y*x + (j)*x + (i+1)] * 6 +
            src[(k)*y*x + (j+1)*x + (i-1)] * -3 + src[(k)*y*x + (j+1)*x + (i+1)] * 3;

        _s_x = src[(k+1)*y*x + (j-1)*x + (i-1)] * -1 + src[(k+1)*y*x + (j-1)*x + (i+1)] +
            src[(k+1)*y*x + (j)*x + (i-1)] * -3 + src[(k+1)*y*x + (j)*x + (i+1)] * 3 +
            src[(k+1)*y*x + (j+1)*x + (i-1)] * -1 + src[(k+1)*y*x + (j+1)*x + (i+1)];
        
        s_z = _s_x + _s_y + _s_z;

        temp = sqrtf(powf((float)s_x, 2) + powf((float)s_y, 2) + powf((float)s_z, 2));
        int des_index = k*y*x + j*x + i;
        if(abs(temp)>1e-5){
            des[des_index] = src[des_index];
        } else {
            des[k*y*x + j*x + i] = 0;
        }
        //des[k*y*x + j*x + i] = (short)temp;
    }

}

hipError_t convolution3DCUDA(
    const short* src,
    short* des,
    const int x,
    const int y,
    const int z
) {
    int array_size = x * y * z;
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    short *src_gpu, *des_gpu;

#ifdef MONITOR_TIME
    clock_t start, end;
    start = clock();
#endif

    cudaStatus = hipMalloc((void**) &src_gpu, sizeof(short) * array_size); 
    cudaStatus = hipMalloc((void**) &des_gpu, sizeof(short) * array_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //hipMemcpy2D(ac, sizeof(float) * n, a, sizeof(float) * lda, sizeof(float) * n, n, hipMemcpyHostToDevice);

    cudaStatus = hipMemcpy(src_gpu, src, array_size*sizeof(short), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //unsigned int num_thread = (array_size -1)% NUM_THREADS + 1;
    //unsigned int blocks = (array_size + num_thread - 1) / num_thread;
    //doConvolutionInPointCUDA<<<blocks, num_thread>>>(src_gpu, des_gpu, m, n, num_thread);

    unsigned int blocks = (y*z + NUM_THREADS - 1) / NUM_THREADS;
    doConvolutionInLineCUDA<<<blocks, NUM_THREADS>>>(src_gpu, des_gpu, x, y, z);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(des, des_gpu, array_size*sizeof(short), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n, sizeof(float) * n, n, hipMemcpyDeviceToHost);
#ifdef MONITOR_TIME
    end = clock();
    float cost_time = (float)(end - start) / CLOCKS_PER_SEC;
    printf("The cost time is: %f\n", cost_time);
#endif

Error:
    hipFree(src_gpu);
    hipFree(des_gpu);
    return cudaStatus;
}

int MY_EXPORT Get3DBorder(short* src, short* des, const int x , const int y, const int z) {
    if(x*y*z <= 0){
        return 0;
    }
    if(src == NULL || des == NULL){
        fprintf(stderr, "error parameters!");
        return 1;
    }

    InitCUDA();
    hipError_t cudaStatus;

    cudaStatus = convolution3DCUDA(src, des, x , y, z);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

#ifdef DUMP_DEBUG
    MyDumpBuffer(L"e:\\c_in_3d", (src+x*y*0), x*y);
    MyDumpBuffer(L"e:\\c_out_3d", (des+x*y*0), x*y);
#endif

    return 0;
}