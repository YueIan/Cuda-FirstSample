#include "hip/hip_runtime.h"
#include<math.h>
#include "sobel.cuh"
extern bool InitCUDA();

#define MONITOR_TIME
#define NUM_THREADS 256

 /*
    int matrix_gx[mn] = {
        -1, 0, 1,
        -2, 0, 2,
        -1, 0, 1
    };

    int matrix_gy[mn] = {
        -1, -2, -1,
        0, 0, 0,
        1, 2, 1
    };
*/

__global__ void doConvolutionInPointCUDA(
    const float* src,
    float* des,
    const int m,
    const int n,
    const int thread_count
) {
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;
    int position = (block_id-1)*thread_count + (thread_id-1);
    int x = position % m;
    int y = position / m;
    if(x <= 0 || x >= m-1 || y <= 0 || y >= n-1){
        return;
    }
   
    float s_x = src[(y+1)*m + (x-1)] + 2 * src[(y+1)*m + (x)] + src[(y+1)*m + (x+1)] - 
        (src[(y-1)*m + (x-1)] + 2 * src[(y-1)*m + (x)] + src[(y-1)*m + (x+1)]);

    float s_y = src[(y-1)*m + (x+1)] + 2 * src[(y)*m + (x+1)] + src[(y+1)*m + (x+1)] - 
        (src[(y-1)*m + (x-1)] + 2 * src[(y)*m + (x-1)] + src[(y+1)*m + (x-1)]);

    float temp = sqrtf(powf(s_x, 2) + powf(s_y, 2));
    if(abs(temp)>1e-5){
        des[position] = 255.f;
    } else {
        des[position] = 0.f;
    }
    //des[position] = temp;
}

/*
every thread calculate one row
*/
__global__ void doConvolutionInLineCUDA(
    const float* src,
    float* des,
    const int m,
    const int n
) {
    const int thread_id = threadIdx.x;
    const int block_id = blockIdx.x;
    int x = 0;
    int y = block_id*NUM_THREADS + thread_id;

    if(y <= 0 || y >= n-1){
        return;
    }

    for(x; x<m; x++){
        if(x <= 0 || x >= m-1){
            continue;
        }
   
        float s_x = src[(y+1)*m + (x-1)] + 2 * src[(y+1)*m + (x)] + src[(y+1)*m + (x+1)] - 
            (src[(y-1)*m + (x-1)] + 2 * src[(y-1)*m + (x)] + src[(y-1)*m + (x+1)]);

        float s_y = src[(y-1)*m + (x+1)] + 2 * src[(y)*m + (x+1)] + src[(y+1)*m + (x+1)] - 
            (src[(y-1)*m + (x-1)] + 2 * src[(y)*m + (x-1)] + src[(y+1)*m + (x-1)]);

        float temp = sqrtf(powf(s_x, 2) + powf(s_y, 2));
        if(abs(temp)>1e-5){
            des[y*m+x] = src[y*m+x];
        } else {
            des[y*m+x] = 0.f;
        }
        //des[y*m+x] = temp;
    }

}

hipError_t convolutionCUDA(
    const float* src,
    float* des,
    const int m,
    const int n
) {
    int array_size = m * n;
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    float *src_gpu, *des_gpu;

#ifdef MONITOR_TIME
    clock_t start, end;
    start = clock();
#endif

    cudaStatus = hipMalloc((void**) &src_gpu, sizeof(float) * array_size); 
    cudaStatus = hipMalloc((void**) &des_gpu, sizeof(float) * array_size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
{
    //hipMemcpy2D(ac, sizeof(float) * n, a, sizeof(float) * lda, sizeof(float) * n, n, hipMemcpyHostToDevice);

    cudaStatus = hipMemcpy(src_gpu, src, array_size*sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //unsigned int num_thread = (array_size -1)% NUM_THREADS + 1;
    //unsigned int blocks = (array_size + num_thread - 1) / num_thread;
    //doConvolutionInPointCUDA<<<blocks, num_thread>>>(src_gpu, des_gpu, m, n, num_thread);

    unsigned int blocks = (n + NUM_THREADS - 1) / NUM_THREADS;
    doConvolutionInLineCUDA<<<blocks, n>>>(src_gpu, des_gpu, m, n);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(des, des_gpu, array_size*sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * n, sizeof(float) * n, n, hipMemcpyDeviceToHost);
#ifdef MONITOR_TIME
    end = clock();
    float cost_time = (float)(end - start) / CLOCKS_PER_SEC;
    printf("The cost time is: %f\n", cost_time);
#endif
}
Error:
    hipFree(src_gpu);
    hipFree(des_gpu);
    return cudaStatus;
}

void doConvolution(
    const float* src,
    float* des,
    const int m,
    const int n
) {
    
    for(int y=0;y<n;y++){
        for(int x=0;x<m;x++){
            if(y <= 0 || y >= n-1 || x <= 0 || x >= m-1){
                continue;
            }
            float s_x = src[(y+1)*m + (x-1)] + 2 * src[(y+1)*m + (x)] + src[(y+1)*m + (x+1)] - 
            (src[(y-1)*m + (x-1)] + 2 * src[(y-1)*m + (x)] + src[(y-1)*m + (x+1)]);

            float s_y = src[(y-1)*m + (x+1)] + 2 * src[(y)*m + (x+1)] + src[(y+1)*m + (x+1)] - 
                (src[(y-1)*m + (x-1)] + 2 * src[(y)*m + (x-1)] + src[(y+1)*m + (x-1)]);

            float temp = sqrtf(powf(s_x, 2) + powf(s_y, 2));
            if(abs(temp)>1e-5){
                des[y*m+x] = src[y*m+x];
            } else {
                des[y*m+x] = 0.f;
            }
        }
    }

}

int MY_EXPORT Get2DBorder(float* src, float* des, const int m , const int n) {
    if(m*n <= 0){
        return 0;
    }
    if(src == NULL || des == NULL){
        fprintf(stderr, "error parameters!");
        return 1;
    }

    bool has_cuda = InitCUDA();
    if(has_cuda){
        hipError_t cudaStatus;

        cudaStatus = convolutionCUDA(src, des, m ,n);

        // hipDeviceReset must be called before exiting in order for profiling and
        // tracing tools such as Nsight and Visual Profiler to show complete traces.
        //cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceReset failed!");
            return 1;
        }
    } else {
        doConvolution(src, des, m ,n);
    }

    return 0;
}